
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include <nvToolsExt.h>
#include "cuda_kernel_launcher.h"
#include <cuda_runtime.h>
#include <cstdio>
#include <LightGBM/utils/log.h>

using namespace LightGBM;

void cuda_histogram(
                data_size_t     leaf_num_data,
                data_size_t     num_data,
                bool            use_all_features,
                bool            is_constant_hessian,
                int             num_workgroups,
                cudaStream_t    stream,
                uint8_t*        arg0,
                uint8_t*        arg1,
                data_size_t     arg2,
                data_size_t*    arg3,
                data_size_t     arg4,
                score_t*        arg5,
                score_t*        arg6,
                score_t         arg6_const,
                char*           arg7,
                volatile int*   arg8,
                void*		arg9,
                size_t          exp_workgroups_per_feature) {

 if (leaf_num_data == num_data) {
   //Log::Debug("calling histogram_fulldata_kernel. exp_workgroups_per_feature=%d", exp_workgroups_per_feature);
   
   if (!is_constant_hessian) { 
     histogram256_fulldata<<<num_workgroups, 256, 0, stream>>>(
       arg0,
       arg1,
       arg2,
       reinterpret_cast<const uint*>(arg3),
       arg4,
       arg5,
       static_cast<float*>(arg6),
       arg7,
       arg8,
       static_cast<acc_type*>(arg9),
       exp_workgroups_per_feature);
   }
   else { 
     histogram256_fulldata<<<num_workgroups, 256, 0, stream>>>(
       arg0,
       arg1,
       arg2,
       reinterpret_cast<const uint*>(arg3),
       arg4,
       arg5,
       arg6_const, 
       arg7,
       arg8,
       static_cast<acc_type*>(arg9),
       exp_workgroups_per_feature);
   }

 }
 else {
   if (use_all_features) {
     // seems all features is always enabled, so this should be the same as fulldata
     //Log::Debug("calling histogram256 instead of allfeats. exp_workgroups_per_feature=%d", exp_workgroups_per_feature);
     
     if (!is_constant_hessian) { 
       histogram256<<<num_workgroups, 256, 0, stream>>>(
         arg0,
         arg1,
         arg2,
         reinterpret_cast<const uint*>(arg3),
         arg4,
         arg5,
         static_cast<float*>(arg6),
         arg7,
         arg8,
         static_cast<acc_type*>(arg9),
         exp_workgroups_per_feature
       );
     }
     else { 
       histogram256<<<num_workgroups, 256, 0, stream>>>(
         arg0,
         arg1,
         arg2,
         reinterpret_cast<const uint*>(arg3),
         arg4,
         arg5,
         arg6_const, 
         arg7,
         arg8,
         static_cast<acc_type*>(arg9),
         exp_workgroups_per_feature
       );
     } 
   }
   else {
     //Log::Debug("calling histogram256. exp_workgroups_per_feature=%d", exp_workgroups_per_feature);
     if (!is_constant_hessian) { 
       histogram256<<<num_workgroups, 256, 0, stream>>>(
         arg0,
         arg1,
         arg2,
         reinterpret_cast<const uint*>(arg3),
         arg4,
         arg5,
         static_cast<float*>(arg6),
         arg7,
         arg8,
         static_cast<acc_type*>(arg9),
         exp_workgroups_per_feature
       );
     }
     else { 
       histogram256<<<num_workgroups, 256, 0, stream>>>(
         arg0,
         arg1,
         arg2,
         reinterpret_cast<const uint*>(arg3),
         arg4,
         arg5,
         arg6_const, 
         arg7,
         arg8,
         static_cast<acc_type*>(arg9),
         exp_workgroups_per_feature
       );
     }
   }
 }
}

#endif // USE_CUDA
